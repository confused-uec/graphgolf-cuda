#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaASPLbox.hpp"
#include "box.hpp"
#include <bitset>
#include <cassert>
namespace graphgolf{

    __global__ void kernel_aspl_box_init(uint *bits, int3 N, int width, int3 M, int* sum){
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        int Ns=N.x*N.y*N.z;
        int Ms=M.x*M.y*M.z;
        if(id>=Ns*width) return;
        int v=id>>(__ffs(width)-1);
        int b=id&(width-1);
        int tmp;
        if(Ms>=(b+1)<<5){
            tmp=0;
        }else if(b<<5>=Ms){
            tmp=0xFFFFFFFF;
        }else{
            tmp=~((1<<(Ms-(b<<5)))-1);
        }
        //int x=v/N.y,y=v%N.y;
        int x=v/(N.y*N.z);
        int y=(v-x*N.y*N.z)/N.z;
        int z=(v-x*N.y*N.z)%N.z;
        if(x<M.x&&y<M.y&&z<M.z){
            int mv=x*M.y*M.z+y*M.z+z;
            if((b<<5)<=mv&&mv<((b+1)<<5)){
                tmp|=1<<(mv-(b<<5));
            }
        }
        sum[id]=0;
        bits[id]=tmp;
    }

    __global__ void kernel_aspl_box_conv(uint *bits, uint *diff_bits, int *edges, int3 N, int width, int3 M, int degree){
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        if(id>=N.x*N.y*N.z*width) return;
        int v=id>>(__ffs(width)-1);
        int b=id&(width-1);
        uint tmp=0;
        int x=v/(N.y*N.z);
        int y=(v-x*N.y*N.z)/N.z;
        int z=(v-x*N.y*N.z)%N.z;
        int xm=x%M.x, ym=y%M.y, zm=z%M.z;
        for(int i=0;i<degree;i++){
            //int to = (N+v+edges[vm*degree+i])%N;
            int diff_x = edges[((xm*M.y*M.z+ym*M.z+zm)*degree+i)*3];
            int diff_y = edges[((xm*M.y*M.z+ym*M.z+zm)*degree+i)*3+1];
            int diff_z = edges[((xm*M.y*M.z+ym*M.z+zm)*degree+i)*3+2];
            int to_x = (N.x+x+diff_x)%N.x;
            int to_y = (N.y+y+diff_y)%N.y;
            int to_z = (N.z+z+diff_z)%N.z;
            int to_v = to_x*N.y*N.z+to_y*N.z+to_z;
            tmp|=bits[(to_v<<(__ffs(width)-1))+b];
        }
        tmp&=~bits[id];
        diff_bits[id]=tmp;
    }

    __global__ void kernel_aspl_box_apply(uint *bits, uint *diff_bits, int *sum, int N, int width, int step){
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        if(id>=N*width) return;
        uint tmp=diff_bits[id];
        bits[id]|=tmp;
        sum[id]+=__popc(tmp)*step;
    }

    //1024 -> 1
    __global__ void kernel_aspl_box_reduce_plus(int *sum, int64_t *ret, int length){
        __shared__ int64_t tmp[32];
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        int64_t elem = id<length?sum[id]:0;
        #pragma unroll
        for(int delta=16;delta;delta>>=1){
            elem+=__shfl_down_sync(0xFFFFFFFF, elem, delta);
        }
        int laneid=threadIdx.x&0x1F;
        int warpid=threadIdx.x>>5;
        if(laneid==0) tmp[warpid]=elem;
        __syncthreads();
        if(warpid) return;
        elem=tmp[laneid];
        #pragma unroll
        for(int delta=16;delta;delta>>=1){
            elem+=__shfl_down_sync(0xFFFFFFFF, elem, delta);
        }
        if(threadIdx.x==0) ret[blockIdx.x]=elem;
    }

    __global__ void kernel_aspl_box_reduce_OR(uint *bits, uint *ret, int length){
        __shared__ uint tmp[32];
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        uint elem = id<length?bits[id]:0;
        #pragma unroll
        for(int delta=16;delta;delta>>=1){
            elem|=__shfl_down_sync(0xFFFFFFFF, elem, delta);
        }
        int laneid=threadIdx.x&0x1F;
        int warpid=threadIdx.x>>5;
        if(laneid==0) tmp[warpid]=elem;
        __syncthreads();
        if(warpid) return;
        elem=tmp[laneid];
        #pragma unroll
        for(int delta=16;delta;delta>>=1){
            elem|=__shfl_down_sync(0xFFFFFFFF, elem, delta);
        }
        if(threadIdx.x==0) ret[blockIdx.x]=elem;
    }

    __global__ void kernel_aspl_box_reduce_AND(uint *bits, uint *ret, int length){
        __shared__ uint tmp[32];
        int id = blockIdx.x*blockDim.x+threadIdx.x;
        uint elem = id<length?bits[id]:0xFFFFFFFF;
        #pragma unroll
        for(int delta=16;delta;delta>>=1){
            elem&=__shfl_down_sync(0xFFFFFFFF, elem, delta);
        }
        int laneid=threadIdx.x&0x1F;
        int warpid=threadIdx.x>>5;
        if(laneid==0) tmp[warpid]=elem;
        __syncthreads();
        if(warpid) return;
        elem=tmp[laneid];
        #pragma unroll
        for(int delta=16;delta;delta>>=1){
            elem&=__shfl_down_sync(0xFFFFFFFF, elem, delta);
        }
        if(threadIdx.x==0) ret[blockIdx.x]=elem;
    }

    cudaASPLbox::cudaASPLbox(int Nx, int Ny, int Nz, int Mx, int My, int Mz, int degree, int device=0):Nx(Nx),Ny(Ny),Nz(Nz),Mx(Mx),My(My),Mz(Mz),degree(degree),device(device){
        int N=Nx*Ny*Nz;
        int M=Mx*My*Mz;
        for(width=1;(width<<5)<M;width<<=1);
        nBlock=(N*width+1023)/1024;
        std::cout<<"width: "<<width<<" nBlock: "<<nBlock<<std::endl;
        hipSetDevice(device);
        hipMalloc((void**)&d_bits,N*width*sizeof(uint));
        hipMalloc((void**)&d_diff_bits,N*width*sizeof(uint));
        hipMalloc((void**)&d_sum,N*width*sizeof(int));
        hipHostMalloc((void**)&h_bits,N*width*sizeof(uint));
        hipMalloc((void**)&d_edges,M*degree*sizeof(int)*3);
        hipHostMalloc((void**)&h_edges,M*degree*sizeof(int)*3);
        hipMalloc((void**)&d_ret,nBlock*sizeof(int64_t));
        hipHostMalloc((void**)&h_ret,nBlock*sizeof(int64_t));
        hipMalloc((void**)&d_ret_bits,nBlock*sizeof(uint));
        hipHostMalloc((void**)&h_ret_bits,nBlock*sizeof(uint));
    }
    cudaASPLbox::~cudaASPLbox(){
        hipSetDevice(device);
        hipFree(d_bits);
        hipFree(d_diff_bits);
        hipFree(d_sum);
        hipHostFree(h_bits);
        hipFree(d_edges);
        hipHostFree(h_edges);
        hipFree(d_ret);
        hipHostFree(h_ret);
        hipFree(d_ret_bits);
        hipHostFree(h_ret_bits);
    }
    std::pair<int,double> cudaASPLbox::diameterASPL(box &p){
        hipSetDevice(device);
        int N=Nx*Ny*Nz;
        int M=Mx*My*Mz;
        for(int x=0;x<Mx;x++){
            for(int y=0;y<My;y++){
                for(int z=0;z<Mz;z++){
                    for(int e=0;e<degree;e++){
                        int dx=0,dy=0,dz=0;
                        if(e<p.edges[x][y][z].size()){
                            std::tie(dx,dy,dz)=p.edges[x][y][z][e];
                        }
                        h_edges[((x*My*Mz+y*Mz+z)*degree+e)*3]=dx;
                        h_edges[((x*My*Mz+y*Mz+z)*degree+e)*3+1]=dy;
                        h_edges[((x*My*Mz+y*Mz+z)*degree+e)*3+2]=dz;
                    }
                }
            }
        }
        hipMemcpy(d_edges,h_edges,M*degree*sizeof(int)*3,hipMemcpyHostToDevice);
        //15625x256 = 4x1,000,000
        kernel_aspl_box_init<<<nBlock,1024>>>(d_bits,make_int3(Nx,Ny,Nz),width,make_int3(Mx,My,Mz),d_sum);
        //std::cout<<"N: "<<N<<" M: "<<M<<" width: "<<width<<" nBlock: "<<nBlock<<std::endl;
        hipDeviceSynchronize();
        int diameter=100000000;
        for(int step=1;step<100;step++){
            kernel_aspl_box_conv<<<nBlock,1024>>>(d_bits,d_diff_bits,d_edges,make_int3(Nx,Ny,Nz),width,make_int3(Mx,My,Mz),degree);
            hipDeviceSynchronize();
            kernel_aspl_box_apply<<<nBlock,1024>>>(d_bits,d_diff_bits,d_sum,N,width,step);
            hipDeviceSynchronize();
            kernel_aspl_box_reduce_OR<<<nBlock,1024>>>(d_diff_bits,d_ret_bits,N*width);
            hipDeviceSynchronize();
            hipMemcpy(h_ret_bits,d_ret_bits,nBlock*sizeof(uint),hipMemcpyDeviceToHost);
            uint flag = 0;
            for(int i=0;i<nBlock;i++) flag|=h_ret_bits[i];
            if(flag==0){
                kernel_aspl_box_reduce_AND<<<nBlock,1024>>>(d_bits,d_ret_bits,N*width);
                hipDeviceSynchronize();
                hipMemcpy(h_ret_bits,d_ret_bits,nBlock*sizeof(uint),hipMemcpyDeviceToHost);
                flag=0xFFFFFFFF;
                for(int i=0;i<nBlock;i++) flag&=h_ret_bits[i];
                if(flag==0xFFFFFFFF){
                    //std::cout<<"Diameter: "<<step-1<<std::endl;
                    diameter=step-1;
                }else{
                    std::cout<<"Graph is unconnected!"<<std::endl;
                    return std::make_pair(diameter,100000000.0);
                }
                break;
            }else if(step==200){
                std::cout<<"Too Large Diameter!"<<std::endl;
                return std::make_pair(diameter,100000000.0);
            }
        }
        kernel_aspl_box_reduce_plus<<<nBlock,1024>>>(d_sum,d_ret,N*width);
        hipMemcpy(h_ret,d_ret,nBlock*sizeof(int64_t),hipMemcpyDeviceToHost);
        int64_t total=0;
        for(int i=0;i<nBlock;i++) total+=h_ret[i];
        total*=N/M;
        return std::make_pair(diameter,double(total)/(int64_t(N)*(N-1)));
    }
}